#include "hip/hip_runtime.h"
// Copyright (c) 2020, 2021, NECSTLab, Politecnico di Milano. All rights reserved.

// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions
// are met:
//  * Redistributions of source code must retain the above copyright
//    notice, this list of conditions and the following disclaimer.
//  * Redistributions in binary form must reproduce the above copyright
//    notice, this list of conditions and the following disclaimer in the
//    documentation and/or other materials provided with the distribution.
//  * Neither the name of NECSTLab nor the names of its
//    contributors may be used to endorse or promote products derived
//    from this software without specific prior written permission.
//  * Neither the name of Politecnico di Milano nor the names of its
//    contributors may be used to endorse or promote products derived
//    from this software without specific prior written permission.

// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
// EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
// PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
// CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
// EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
// PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
// OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
// (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

#include <sstream>
#include <assert.h>
#include "personalized_pagerank.cuh"

namespace chrono = std::chrono;
using clock_type = chrono::high_resolution_clock;

//////////////////////////////
//////////////////////////////

// Write GPU kernel here!

__global__ void gpu_calculate_ppr_0(
    int *cols_idx, 
    int* ptr, 
    double* val,
    double* p,
    int* dangling,
    double* result,
    int pers_ver,
    double alpha,
    int V)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;   
    int start = ptr[idx];
    int end = ptr[idx + 1];

    double prod_fact = 0, dang_fact = 0, pers_fact = 0;

    for (int i = start; i < end; i++) {
        prod_fact += val[i] * p[cols_idx[i]];        
    }

    for (int i = 0; i < V; i++){
        dang_fact += dangling[i] * p[i];
    }

    prod_fact *= alpha;
    dang_fact *= alpha / V;
    if (pers_ver == idx)//for the future preprocess pers_ver in a vector check condition
        pers_fact = (1 - alpha);
    
    //__syncthreads();    atomicAdd(res, sum);  

    result[idx] = prod_fact + dang_fact + pers_fact;   
}

//////////////////////////////
//////////////////////////////

// CPU Utility functions;

// Read the input graph and initialize it;
void PersonalizedPageRank::initialize_graph() {
    // Read the graph from an MTX file;
    int num_rows = 0;
    int num_columns = 0;
    read_mtx(graph_file_path.c_str(), &x, &y, &val,
        &num_rows, &num_columns, &E, // Store the number of vertices (row and columns must be the same value), and edges;
        true,                        // If true, read edges TRANSPOSED, i.e. edge (2, 3) is loaded as (3, 2). We set this true as it simplifies the PPR computation;
        false,                       // If true, read the third column of the matrix file. If false, set all values to 1 (this is what you want when reading a graph topology);
        debug,                 
        false,                       // MTX files use indices starting from 1. If for whatever reason your MTX files uses indices that start from 0, set zero_indexed_file=true;
        true                         // If true, sort the edges in (x, y) order. If you have a sorted MTX file, turn this to false to make loading faster;
    );
    if (num_rows != num_columns) {
        if (debug) std::cout << "error, the matrix is not squared, rows=" << num_rows << ", columns=" << num_columns << std::endl;
        exit(-1);
    } else {
        V = num_rows;
    }
    if (debug) std::cout << "loaded graph, |V|=" << V << ", |E|=" << E << std::endl;

    // Compute the dangling vector. A vertex is not dangling if it has at least 1 outgoing edge;
    dangling.resize(V);
    std::fill(dangling.begin(), dangling.end(), 1);  // Initially assume all vertices to be dangling;
    for (int i = 0; i < E; i++) {
        // Ignore self-loops, a vertex is still dangling if it has only self-loops;
        if (x[i] != y[i]) dangling[y[i]] = 0;
    }
    // Initialize the CPU PageRank vector;
    pr.resize(V);
    pr_golden.resize(V);
    // Initialize the value vector of the graph (1 / outdegree of each vertex).
    // Count how many edges start in each vertex (here, the source vertex is y as the matrix is transposed);
    int *outdegree = (int *) calloc(V, sizeof(int));
    for (int i = 0; i < E; i++) {
        outdegree[y[i]]++;
    }
    // Divide each edge value by the outdegree of the source vertex;
    for (int i = 0; i < E; i++) {
        val[i] = 1.0 / outdegree[y[i]];  
    }
    free(outdegree);
}

//convert COO in CSR
void PersonalizedPageRank::converter(){
    std::vector<int> xPtr;
    int ptr=0,previousX;

    // Matrix:
    // 10 20  0  0  0  0
    //  0 30  0 40  0  0
    //  0  0 50 60 70  0
    //  0  0  0  0  0 80

    // coo data:
    //double coo_val[nnz] = { 10.0, 20.0, 30.0, 40.0, 50.0, 60.0, 70.0, 80.0 };
    //int    coo_x[nnz] = { 0, 0, 1, 1, 2, 2, 2, 3 };
    //int    coo_col[nnz] = { 0, 1, 1, 3, 2, 3, 4, 5 };

    // Expected output:
    // csr_val: 10 20 30 40 50 60 70 80
    // csr_col:  0  1  1  3  2  3  4  5
    // csr_x:  0  2  4  7  8

    if(E==0)
        return;
    
    previousX = 0;
    xPtr.push_back(0);

    for (int i =0; i< E; i++) {
        
        while(x[i]!=previousX){
            xPtr.push_back(ptr);
            previousX++;
        }
        ptr++;
    }
    
    for (int i =0; i< V-x[E-1]; i++) {
        xPtr.push_back(ptr); 
    }


    //if (debug){
    //    std::cout << "vettore ptr: ";
    //    for (int i =0; i< xPtr.size(); i++){
    //        std::cout << xPtr[i] << " ";
    //    }
    //    std::cout << "\n";
    //}
    
    x=xPtr;

}

void PersonalizedPageRank::alloc_to_gpu() {
    
    hipMalloc(&d_x, sizeof(double) * x.size());
    hipMalloc(&d_y, sizeof(double) * y.size());
    hipMalloc(&d_val, sizeof(double) * val.size());
    hipMalloc(&d_dangling, sizeof(double) * dangling.size());
    hipMalloc(&d_pr, sizeof(double) * V);
    hipMalloc(&d_newPr, sizeof(double) * V);

    hipMemcpy(d_x, &x[0], sizeof(double) * x.size(), hipMemcpyHostToDevice);
    hipMemcpy(d_y, &y[0], sizeof(double) *  y.size(), hipMemcpyHostToDevice);
    hipMemcpy(d_val, &val[0], sizeof(double) *  val.size(), hipMemcpyHostToDevice);
    hipMemcpy(d_dangling, &dangling[0], sizeof(double) * dangling.size(), hipMemcpyHostToDevice);
    
}


//////////////////////////////
//////////////////////////////

// Allocate data on the CPU and GPU;
void PersonalizedPageRank::alloc() {
    // Load the input graph and preprocess it;
    initialize_graph();
    
    //convert COO in CSR
    converter();

    // Compute the number of blocks for implementations where the value is a function of the input size;
    BlockNum = (V + block_size - 1) / block_size;

    // Allocate any GPU data here;
    alloc_to_gpu();
}

// Initialize data;
void PersonalizedPageRank::init() {
    // Do any additional CPU or GPU setup here;
    // TODO!
}

// Reset the state of the computation after every iteration.
// Reset the result, and transfer data to the GPU if necessary;
void PersonalizedPageRank::reset() {
    // Do any GPU reset here, and also transfer data to the GPU;

    // Reset the PageRank vector (uniform initialization, 1 / V for each vertex);
    pr.clear();
    for (int i=0; i<V;i++){
        pr.push_back(1.0 / V);
    }
    
    
    // Generate a new personalization vertex for this iteration;
    personalization_vertex = rand() % V; 
    if (debug) std::cout << "personalization vertex=" << personalization_vertex << std::endl;

    // Reset the result in GPU and Transfer data to the GPU (hipMemset(d_pr, 1.0 / V, sizeof(double) * V));
    //if it's so stupid we don't need to copy but just set it or even find a way to begin without passing thisdata
    hipMemcpy(d_pr, &pr[0], sizeof(double) * V, hipMemcpyHostToDevice);
    
}

void PersonalizedPageRank::personalized_page_rank_0(int iter){
    auto start_tmp = clock_type::now();
    double *d_temp;
    bool converged = false;

    int = 0;
    while ((!converged && i < max_iterations) || i == 30) {
        // Call the GPU computation.
        gpu_calculate_ppr_0<<<1, 17>>>(d_y, d_x, d_val, d_pr, d_dangling, d_newPr, personalization_vertex, alpha, V);
        
        d_temp=d_pr;
        d_pr=d_newPr;
        d_newPr=d_temp;

        //ensure entire pr is calculated
        hipDeviceSynchronize();

        double err = euclidean_distance(d_pr, d_newPr, V);
        converged = err <= convergence_threshold;
        i++;
    }
    

    // Print performance of GPU, not accounting for transfer time;
    if (debug) {
        // Synchronize computation by hand to measure GPU exec. time;
        hipDeviceSynchronize();
        auto end_tmp = clock_type::now();
        auto exec_time = chrono::duration_cast<chrono::microseconds>(end_tmp - start_tmp).count();
        std::cout << "  pure GPU execution(" << iter << ")=" << double(exec_time) / 1000 << " ms, " << std::endl;
    }

    // Copy the result from the GPU to the CPU;
    //for the future try order values in GPU and trasfer only first 20 
    hipMemcpy(&pr[0], d_pr, sizeof(double) * V, hipMemcpyDeviceToHost);
}

double PersonalizedPageRank::euclidean_distance(double *x, double *y, int N) {
    double result = 0;
    for (int i = 0; i < N; i++) {
        double tmp = x[i] - y[i];
        result += tmp * tmp;
    }
    return std::sqrt(result);
}

// Do the GPU computation here, and also transfer results to the CPU;
void PersonalizedPageRank::execute(int iter) {

    switch (implementation)
    {
    case 0:
        personalized_page_rank_0(iter);
        break;
    
    default:
        break;
    }    
    
}

void PersonalizedPageRank::cpu_validation(int iter) {

    // Reset the CPU PageRank vector (uniform initialization, 1 / V for each vertex);
    std::fill(pr_golden.begin(), pr_golden.end(), 1.0 / V);

    // Do Personalized PageRank on CPU;
    auto start_tmp = clock_type::now();
    personalized_pagerank_cpu(x.data(), y.data(), val.data(), V, E, pr_golden.data(), dangling.data(), personalization_vertex, alpha, 1e-6, 100);
    auto end_tmp = clock_type::now();
    auto exec_time = chrono::duration_cast<chrono::microseconds>(end_tmp - start_tmp).count();
    std::cout << "exec time CPU=" << double(exec_time) / 1000 << " ms" << std::endl;

    // Obtain the vertices with highest PPR value;
    std::vector<std::pair<int, double>> sorted_pr_tuples = sort_pr(pr.data(), V);
    std::vector<std::pair<int, double>> sorted_pr_golden_tuples = sort_pr(pr_golden.data(), V);

    // Check how many of the correct top-20 PPR vertices are retrieved by the GPU;
    std::unordered_set<int> top_pr_indices;
    std::unordered_set<int> top_pr_golden_indices;
    int old_precision = std::cout.precision();
    std::cout.precision(4);
    int topk = std::min(V, topk_vertices);
    for (int i = 0; i < topk; i++) {
        int pr_id_gpu = sorted_pr_tuples[i].first;
        int pr_id_cpu = sorted_pr_golden_tuples[i].first;
        top_pr_indices.insert(pr_id_gpu);
        top_pr_golden_indices.insert(pr_id_cpu);
        if (debug) {
            double pr_val_gpu = sorted_pr_tuples[i].second;
            double pr_val_cpu = sorted_pr_golden_tuples[i].second;
            if (pr_id_gpu != pr_id_cpu) {
                std::cout << "* error in rank! (" << i << ") correct=" << pr_id_cpu << " (val=" << pr_val_cpu << "), found=" << pr_id_gpu << " (val=" << pr_val_gpu << ")" << std::endl;
            } else if (std::abs(sorted_pr_tuples[i].second - sorted_pr_golden_tuples[i].second) > 1e-6) {
                std::cout << "* error in value! (" << i << ") correct=" << pr_id_cpu << " (val=" << pr_val_cpu << "), found=" << pr_id_gpu << " (val=" << pr_val_gpu << ")" << std::endl;
            }
        }
    }
    std::cout.precision(old_precision);
    // Set intersection to find correctly retrieved vertices;
    std::vector<int> correctly_retrieved_vertices;
    set_intersection(top_pr_indices.begin(), top_pr_indices.end(), top_pr_golden_indices.begin(), top_pr_golden_indices.end(), std::back_inserter(correctly_retrieved_vertices));
    precision = double(correctly_retrieved_vertices.size()) / topk;
    if (debug) std::cout << "correctly retrived top-" << topk << " vertices=" << correctly_retrieved_vertices.size() << " (" << 100 * precision << "%)" << std::endl;
}

std::string PersonalizedPageRank::print_result(bool short_form) {
    if (short_form) {
        return std::to_string(precision);
    } else {
        // Print the first few PageRank values (not sorted);
        std::ostringstream out;
        out.precision(3);
        out << "[";
        for (int i = 0; i < std::min(20, V); i++) {
            out << pr[i] << ", ";
        }
        out << "...]";
        return out.str();
    }
}

void PersonalizedPageRank::clean() {
    // Delete any GPU data or additional CPU data;
    
    //free(cpu_data);
    hipFree(d_dangling);
    hipFree(d_pr);
    hipFree(d_newPr);
    hipFree(d_val);
    hipFree(d_x);
    hipFree(d_y); 
}
